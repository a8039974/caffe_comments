#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layers/sigmoid_cross_entropy_loss_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {


template <typename Dtype>
__global__ void SigmoidCrossEntropyLossForwardGPU(const int nthreads,
          const Dtype* input_data, const Dtype* target, Dtype* loss,
          const bool has_ignore_label_, const int ignore_label_,
          Dtype* counts) {
  CUDA_KERNEL_LOOP(i, nthreads) {
    const int target_value = static_cast<int>(target[i]);
    if (has_ignore_label_ && target_value == ignore_label_) {
      loss[i] = 0;
      counts[i] = 0;
    } else {
      loss[i] = input_data[i] * (target[i] - (input_data[i] >= 0)) -
          log(1 + exp(input_data[i] - 2 * input_data[i] *
          (input_data[i] >= 0)));
      counts[i] = 1;
    }
  }
}

template <typename Dtype>
__global__ void SigmoidCrossEntropyLossIgnoreDiffGPU(const int count,
    const int ignore_label, const Dtype* target, Dtype* diff) {
  CUDA_KERNEL_LOOP(i, count) {
    const int target_value = static_cast<int>(target[i]);
    if (target_value == ignore_label) {
      diff[i] = 0;
    }
  }
}


template <typename Dtype>
void SigmoidCrossEntropyLossLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  // The forward pass computes the sigmoid outputs.
  // 内嵌的 sigmoid 函数进行前向传播 (放入了 sigmoid_output_ 里)
  sigmoid_bottom_vec_[0] = bottom[0];
  sigmoid_layer_->Forward(sigmoid_bottom_vec_, sigmoid_top_vec_);
  // Compute the loss (negative log likelihood)
  // 获取输入数据的像素点总数
  const int count = bottom[0]->count();
  // Stable version of loss computation from input data
  // 获取只读输入数据和标签
  const Dtype* input_data = bottom[0]->gpu_data();
  const Dtype* target = bottom[1]->gpu_data();
  // Since this memory is not used for anything until it is overwritten
  // on the backward pass, we use it here to avoid having to allocate new GPU
  // memory to accumulate intermediate results in the kernel.
  // 获取可写入输入数据和标签
  Dtype* loss_data = bottom[0]->mutable_gpu_diff();
  Dtype* count_data = bottom[1]->mutable_gpu_diff();
  Dtype valid_count;

  // NOLINT_NEXT_LINE(whitespace/operators)
  // ----- 前向传播计算loss -----
  SigmoidCrossEntropyLossForwardGPU<Dtype><<<CAFFE_GET_BLOCKS(count),
      CAFFE_CUDA_NUM_THREADS>>>(count, input_data, target, loss_data,
      has_ignore_label_, ignore_label_, count_data);
  // Only launch another CUDA kernel if we actually need the valid count.
  // 如果normlize使用VALID模式,且有label无效的情况下, 才去认真地计算有效点数
  if (normalization_ == LossParameter_NormalizationMode_VALID &&
      has_ignore_label_) {
    caffe_gpu_asum(count, count_data, &valid_count);
  } else {
    valid_count = count;
  }
  Dtype loss;

  // 把每一个点的loss加和
  caffe_gpu_asum(count, loss_data, &loss);
  // 归一化loss, 放入top[0]
  normalizer_ = get_normalizer(normalization_, valid_count);
  top[0]->mutable_cpu_data()[0] = loss / normalizer_;
}

template <typename Dtype>
void SigmoidCrossEntropyLossLayer<Dtype>::Backward_gpu(
    const vector<Blob<Dtype>*>& top, const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[1]) {
    LOG(FATAL) << this->type()
               << " Layer cannot backpropagate to label inputs.";
  }
  if (propagate_down[0]) {
    // First, compute the diff
    const int count = bottom[0]->count();
    // 获取内嵌 sigmoid 函数的输出值
    const Dtype* sigmoid_output_data = sigmoid_output_->gpu_data();
    // 获取标签
    const Dtype* target = bottom[1]->gpu_data();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    caffe_copy(count, sigmoid_output_data, bottom_diff);

    // 点对点(element-wise)相减的操作
    caffe_gpu_axpy(count, Dtype(-1), target, bottom_diff);

    // Zero out gradient of ignored targets.
    if (has_ignore_label_) {
      // NOLINT_NEXT_LINE(whitespace/operators)
      SigmoidCrossEntropyLossIgnoreDiffGPU<Dtype><<<CAFFE_GET_BLOCKS(count),
        CAFFE_CUDA_NUM_THREADS>>>(count, ignore_label_, target, bottom_diff);
    }

    // Scale down gradient
    // 归一化 loss 并放入 bottom_diff
    Dtype loss_weight = top[0]->cpu_diff()[0] / normalizer_;
    caffe_gpu_scal(count, loss_weight, bottom_diff);
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(SigmoidCrossEntropyLossLayer);

}  // namespace caffe
